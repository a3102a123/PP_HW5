#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define XBLOCK_SIZE 16
#define YBLOCK_SIZE 12

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int width,int count, int *output, int *test) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    float c_re = lowerX + i * stepX;
    float c_im = lowerY + j * stepY;

    int idx;
    float z_re = c_re, z_im = c_im;
    for (idx = 0; idx < count; ++idx)
    {

        if (z_re * z_re + z_im * z_im > 4.f)
        break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    int index = (j * width + i);
    output[index] = idx;
    test[index] = index;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    // allocate memory in host & device
    int *host_mem, *dev_mem;
    host_mem = (int *)malloc(size * sizeof(int));
    hipMalloc((void **)&dev_mem, size * sizeof(int));
    // debug memory
    int *host_test,*dev_test;
    host_test = (int *)malloc(size * sizeof(int));
    hipMalloc((void **)&dev_test, size * sizeof(int));
    // GPU processing 
    dim3 num_block(resX / XBLOCK_SIZE, resY / YBLOCK_SIZE);
    dim3 block_size(XBLOCK_SIZE, YBLOCK_SIZE);
    mandelKernel<<<num_block, block_size>>>(lowerX, lowerY, stepX, stepY, resX, maxIterations, dev_mem,dev_test);
    hipDeviceSynchronize();
    // GPU translate result data back
    hipMemcpy(host_mem, dev_mem, size * sizeof(int), hipMemcpyDeviceToHost);
    // debug message
    hipMemcpy(host_test, dev_test, size * sizeof(int), hipMemcpyDeviceToHost);
    for(int j = 0 ; j < resY ; j++){
        for(int i = 0 ; i < resX ; i++)
            printf("%d ",host_test[j * resX + i]);
        printf("\n");
    }
    memcpy(img, host_mem, size);
    free(host_mem);
    hipFree(dev_mem);
}
