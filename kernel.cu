#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define XBLOCK_SIZE 16
#define YBLOCK_SIZE 12
#define GROUP_SIZE 2

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,size_t width,int count, int *output, int group_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    for(int k = 0 ; k < group_size ; k++){
        int i = blockIdx.x * blockDim.x + threadIdx.x * group_size + k;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        float c_re = lowerX + i * stepX;
        float c_im = lowerY + j * stepY;

        int idx;
        float z_re = c_re, z_im = c_im;
        for (idx = 0; idx < count; ++idx)
        {

            if (z_re * z_re + z_im * z_im > 4.f)
            break;

            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = c_re + new_re;
            z_im = c_im + new_im;
        }

        int* row = (int*)((char*)output + j * width);
        row[i] = idx;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY * sizeof(int);
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    // allocate memory in host & device
    int *host_mem, *dev_mem;
    size_t pitch;
    hipHostMalloc(&host_mem, size, hipHostMallocDefault);
    hipMallocPitch(&dev_mem, &pitch, resX * sizeof(int), resY);
    // GPU processing 
    dim3 num_block(resX / XBLOCK_SIZE / GROUP_SIZE, resY / YBLOCK_SIZE);
    dim3 block_size(XBLOCK_SIZE, YBLOCK_SIZE);
    mandelKernel<<<num_block, block_size>>>(lowerX, lowerY, stepX, stepY, pitch, maxIterations, dev_mem, GROUP_SIZE);
    hipDeviceSynchronize();
    // GPU translate result data back
    hipMemcpy2D(host_mem, size/resY, dev_mem, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, host_mem, size);
    hipHostFree(host_mem);
    hipFree(dev_mem);
}
