#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define XBLOCK_SIZE 32
#define YBLOCK_SIZE 24

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int width,int count, int *output, int round, int round_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = (j * width + i);
    float c_re = lowerX + i * stepX;
    float c_im = lowerY + (j + round * round_size) * stepY;

    int idx;
    float z_re = c_re, z_im = c_im;
    for (idx = 0; idx < count; ++idx)
    {

        if (z_re * z_re + z_im * z_im > 4.f)
        break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    output[index] = idx;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY * sizeof(int);
    int round_size = resX * YBLOCK_SIZE * sizeof(int);
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    // allocate memory in host & device
    int *host_mem, *host_temp_mem, *dev_ptr, *dev_pre_ptr, *dev_mem1, *dev_mem2,*temp_ptr;
    host_mem = (int *)malloc(size);
    hipHostAlloc(&host_temp_mem, size, hipHostMallocDefault);
    hipMalloc((void **)&dev_mem1, round_size);
    hipMalloc((void **)&dev_mem2, round_size);
    dev_ptr = dev_mem1;
    dev_pre_ptr = dev_mem2;
    // GPU processing 
    dim3 num_block(resX / XBLOCK_SIZE, 1);
    dim3 block_size(XBLOCK_SIZE, YBLOCK_SIZE);
    for(int j = 0 ; j < resY / YBLOCK_SIZE ; j++){
        mandelKernel<<<num_block, block_size>>>(lowerX, lowerY, stepX, stepY, resX, maxIterations, dev_ptr, j, YBLOCK_SIZE);
        hipDeviceSynchronize();
        temp_ptr = dev_ptr;
        dev_ptr = dev_pre_ptr;
        dev_pre_ptr = temp_ptr;
        temp_ptr = host_mem + (round_size / sizeof(int)) * j;
        hipMemcpy( temp_ptr, dev_pre_ptr, round_size, hipMemcpyDeviceToHost);
    }
    /*for(int j = 25 ; j < 28 ; j++){
        for(int i = 0 ; i < resX ; i++)
            printf("%d ",host_mem[i + j * resX]);
        printf("\n");
    }*/
    // GPU translate result data back
    memcpy(img, host_mem, size);
    hipHostFree(host_mem);
    hipFree(dev_mem1);
    hipFree(dev_mem2);
}
