#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define XBLOCK_SIZE 32
#define YBLOCK_SIZE 24

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int width,int count, int *output, int round, int round_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = (j * width + i);
    float c_re = lowerX + i * stepX;
    float c_im = lowerY + j * stepY;

    i += round * round_size; 

    int idx;
    float z_re = c_re, z_im = c_im;
    for (idx = 0; idx < count; ++idx)
    {

        if (z_re * z_re + z_im * z_im > 4.f)
        break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    output[index] = idx;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int size = resX * resY * sizeof(int);
    int round_size = resX / XBLOCK_SIZE * resY / YBLOCK_SIZE;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    // allocate memory in host & device
    int *host_mem, *host_temp_mem, *dev_ptr, *dev_pre_ptr, *dev_mem1, *dev_mem2;
    host_mem = (int *)malloc(size);
    hipHostAlloc(&host_temp_mem, size, hipHostMallocDefault);
    hipMalloc((void **)&dev_mem1, round_size);
    hipMalloc((void **)&dev_mem2, round_size);
    dev_ptr = dev_mem1;
    dev_pre_ptr = dev_mem2;
    // GPU processing 
    dim3 num_block(resX / XBLOCK_SIZE, resY / YBLOCK_SIZE);
    dim3 block_size(1, YBLOCK_SIZE);
    for(int i = 0 ; i < XBLOCK_SIZE ; i++){
        mandelKernel<<<num_block, block_size>>>(lowerX, lowerY, stepX, stepY, resX, maxIterations, dev_mem1, i, round_size);
        hipDeviceSynchronize();
        hipMemcpy(host_mem + i * round_size, dev_mem1, round_size, hipMemcpyDeviceToHost);
    }
    // GPU translate result data back
    memcpy(img, host_mem, size);
    hipHostFree(host_mem);
    hipFree(dev_mem1);
    hipFree(dev_mem2);
}
